#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#ifndef __CUDACC_RTC__   // for __syncthreads()
#define __CUDACC_RTC__
#endif // !(__CUDACC_RTC__)

#ifndef __HIPCC__  // for atomicAdd()
#define __HIPCC__
#endif // !__HIPCC__

#include "device_atomic_functions.h"
#include "hip/device_functions.h"
#include <stdio.h>
#include <iostream>	
#include <time.h>
#include <stdlib.h>
using namespace std;

void print_function(float*, int, int);
void fill_function(float*, int, int, int is_zero = -1);


__global__ void kernel_1(int columns, int rows, float* mat1, float* matanswer) {
	int columna = threadIdx.x;  //En que columna operamos (no filas)
	float temp_value = 0;

	for (int k = 0; k < rows; k++) {
		temp_value = temp_value + mat1[(k * columns) + columna];
	}
	matanswer[columna] = temp_value;
}

__global__ void kernel_2_shared(int columns, int rows, float* mat1, float* matanswer) {
	__shared__ float temp_answer[32];
	if (threadIdx.y == 0) {
		temp_answer[threadIdx.x] = float(0);
	}
	__syncthreads();  //Existe pero no es reconocido por Itellisense

	int thread_mat_colid = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_mat_rowid = blockIdx.y * blockDim.y + threadIdx.y;
	int position_in_matrix = thread_mat_rowid * columns + thread_mat_colid;

	atomicAdd(&(temp_answer[threadIdx.x]), mat1[position_in_matrix]);
	__syncthreads();  //Existe pero no es reconocido por Itellisense

	if (blockIdx.x == 0 && blockIdx.y == 1 && threadIdx.x == 0 && threadIdx.y == 0) {
		printf("%d Value %f \n", position_in_matrix, mat1[position_in_matrix]);
	}

	if (threadIdx.y == rows - 1) {
		atomicAdd(&(matanswer[thread_mat_colid]), temp_answer[threadIdx.x]);
	}
	__syncthreads();
	//printf("BlockID_x: %d BlockID_y: %d Blockdim_x: %d  Blockdim_y: %d ThreadIdx_x: %d ThreadIdx_y: %d \n",  blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
}

void column_sum_facade(int columns, int rows, float* mat1, float* matanswer) {
	int size = columns * rows * sizeof(float); // Matriz de entrada
	int size_2 = columns * sizeof(float);      // Vector con los resultados     
	float* dev_mat1;
	float* dev_matanswer;

	hipSetDevice(0);
	hipMalloc((void**)&dev_mat1, size);       //Reservando espacio
	hipMalloc((void**)&dev_matanswer, size_2);

	hipMemcpy(dev_mat1, mat1, size, hipMemcpyHostToDevice);   //Inicializando con datos
	hipMemset(dev_matanswer, 0, size_2);

	//kernel_1 <<< 1, 1024>>> (columns, rows, dev_mat1, dev_matanswer);
	dim3 grid_dim = dim3(ceil(columns / 32), ceil(rows / 16), 1);  // Numero de bloques en un grid
	dim3 block_dim = dim3(32, 16, 1);                              // Numero de threads en un bloque
	kernel_2_shared << < grid_dim, block_dim >> > (columns, rows, dev_mat1, dev_matanswer);

	hipMemcpy(matanswer, dev_matanswer, size_2, hipMemcpyDeviceToHost);
	hipFree(dev_mat1);
	hipFree(dev_matanswer);
}


// Programando CPU <-- Separando ambos ambientes //
int main() {
	srand(time(NULL));

	const int col_siz = 64;
	const int row_siz = 16;

	float* arr_1 = new float[col_siz * row_siz];
	float* arr_ans = new float[col_siz];


	fill_function(arr_1, col_siz, row_siz);
	fill_function(arr_ans, col_siz, 1, 0);

	column_sum_facade(col_siz, row_siz, arr_1, arr_ans);

	print_function(arr_1, col_siz, row_siz);
	print_function(arr_ans, col_siz, 1);


	hipDeviceReset();
	return 0;
}


void print_function(float* arr, int col, int row) {
	for (int i = 0; i < row; i++) {
		for (int j = 0; j < col; j++) {
			cout << arr[i * col + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

void fill_function(float* arr, int col, int row, int is_zero) {
	if (is_zero == -1) {
		for (int i = 0; i < row; i++) {
			for (int j = 0; j < col; j++) {
				arr[i * col + j] = float(rand() % 10);
			}
		}
	}
	else {
		for (int i = 0; i < row; i++) {
			for (int j = 0; j < col; j++) {
				arr[i * col + j] = float(is_zero);
			}
		}
	}

}
