#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>	
using namespace std;

void print_function(int*, int);

// Programando Tarjeta// 
__global__ void k_vector_sum_constante(int* arr, int* ans, int n, int constant) {  // arr -> Vector, ans -> Resultado, n -> tama�o de array
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		ans[idx] = arr[idx] + constant;
	}
}

__global__ void k_vector_minus_constante(int* arr, int* ans, int n, int constant) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		ans[idx] = arr[idx] - constant;
	}
}

__global__ void k_vector_sum_vector(int *ans, const int *arr_1, const int *arr_2, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		ans[idx] = arr_1[idx] + arr_2[idx];
	}
}

__global__ void k_vector_minus_vector(int *ans, const int *arr_1, const int *arr_2, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		ans[idx] = arr_1[idx] - arr_2[idx];
	}
}


void facade_k_vector_sum_minus_constante(int *arr, int* answer, unsigned int size, int k, char mood) {
	int *dev_arr = 0;
	int *dev_answer = 0;
	int dev_k = k;

	hipSetDevice(0); // Seleccionando primera tarjeta de video en caso de sistemas con multiples tarjetas.
	hipMalloc((void**)&dev_arr, size * sizeof(int)); // Reservando espacio de memoria en dispositivo
	hipMalloc((void**)&dev_answer, size * sizeof(int));

	hipMemcpy(dev_arr, arr, size * sizeof(int), hipMemcpyHostToDevice); // Copiando contenido de vector a dispositivo
	if (mood == 's') {
		k_vector_sum_constante << <ceil(size / 500), 500 >> > (dev_arr, dev_answer, size, dev_k);
	}
	else {
		k_vector_minus_constante << < ceil(size / 500), 500 >> > (dev_arr, dev_answer, size, dev_k);
	}
	hipDeviceSynchronize();

	hipMemcpy(answer, dev_answer, size * sizeof(int), hipMemcpyDeviceToHost); // Devolviendo datos del dispositivo
	hipFree(dev_arr);
	hipFree(dev_answer);
	return;
}

void facade_k_vector_sum_minus_vector(int* ans, int* arr_1, int* arr_2, unsigned int size, char mood) {
	int* dev_ans = 0;
	int* dev_arr_1 = 0;
	int* dev_arr_2 = 0;

	hipSetDevice(0);
	hipMalloc((void**)&dev_ans, size * sizeof(int));
	hipMalloc((void**)&dev_arr_1, size * sizeof(int));
	hipMalloc((void**)&dev_arr_2, size * sizeof(int));

	hipMemcpy(dev_arr_1, arr_1, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr_2, arr_2, size * sizeof(int), hipMemcpyHostToDevice);

	if (mood == 's') {
		k_vector_sum_vector << <ceil(size / 500), 500 >> > (dev_ans, dev_arr_1, dev_arr_2, size);
	}
	else {
		k_vector_minus_vector << <ceil(size / 500), 500 >> > (dev_ans, dev_arr_1, dev_arr_2, size);
	}
	hipDeviceSynchronize();

	hipMemcpy(ans, dev_ans, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_ans);
	hipFree(dev_arr_1);
	hipFree(dev_arr_2);
	return;
}

// Programando CPU <-- Separando ambos ambientes //
int main() {
	int size = 1000;
	int* arr = new int[size];
	int* arr_2 = new int[size];
	int* answer = new int[size];
	int constant = 2; // Cantidad a sumar o restar

	for (int i = 0; i < size; i++) {//Inicializando
		arr[i] = i;
		arr_2[i] = i + i;
		answer[i] = 1;
	}

	//Probando suma resta de vector con vector 
	cout << "Primer vector" << endl;
	print_function(arr, size);
	cout << "Segundo vector" << endl;
	print_function(arr_2, size);
	facade_k_vector_sum_minus_vector(answer, arr, arr_2, size, 's'); // s -> suma, m -> resta
	cout << "Vector resultante" << endl;
	print_function(answer, size);

	hipDeviceReset();
	return 0;
}

void print_function(int* arr, int size) {
	for (int i = 0; i < size; i++) {
		cout << arr[i] << ", ";
	}
	cout << endl << endl;
}


