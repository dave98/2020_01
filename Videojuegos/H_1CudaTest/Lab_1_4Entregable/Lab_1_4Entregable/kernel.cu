#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>	
using namespace std;

void print_function(int*, int);

// Programando Tarjeta// 
__global__ void k_vector_sum_constante(int* arr, int* ans, int n, int constant) {  // arr -> Vector, ans -> Resultado, n -> tama�o de array
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		ans[idx] = arr[idx] + constant;
	}
}

__global__ void k_vector_minus_constante(int* arr, int* ans, int n, int constant) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		ans[idx] = arr[idx] - constant;
	}
}

__global__ void k_vector_sum_vector(int *ans, const int *arr_1, const int *arr_2, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		ans[idx] = arr_1[idx] + arr_2[idx];
	}
}

__global__ void k_vector_minus_vector(int *ans, const int *arr_1, const int *arr_2, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		ans[idx] = arr_1[idx] - arr_2[idx];
	}
}


void facade_k_vector_sum_minus_constante(int *arr, int* answer, unsigned int size, int k, char mood) {
	int *dev_arr = 0;
	int *dev_answer = 0;
	int dev_k = k;

	hipSetDevice(0); // Seleccionando primera tarjeta de video en caso de sistemas con multiples tarjetas.
	hipMalloc((void**)&dev_arr, size * sizeof(int)); // Reservando espacio de memoria en dispositivo
	hipMalloc((void**)&dev_answer, size * sizeof(int));

	hipMemcpy(dev_arr, arr, size * sizeof(int), hipMemcpyHostToDevice); // Copiando contenido de vector a dispositivo
	if (mood == 's') {
		k_vector_sum_constante << <ceil(size / 500), 500 >> > (dev_arr, dev_answer, size, dev_k);
	}
	else {
		k_vector_minus_constante << < ceil(size / 500), 500 >> > (dev_arr, dev_answer, size, dev_k);
	}
	hipDeviceSynchronize();

	hipMemcpy(answer, dev_answer, size * sizeof(int), hipMemcpyDeviceToHost); // Devolviendo datos del dispositivo
	hipFree(dev_arr);
	hipFree(dev_answer);
	return;
}

void facade_k_vector_sum_minus_vector(int* ans, int* arr_1, int* arr_2, unsigned int size, char mood) {
	int* dev_ans = 0;
	int* dev_arr_1 = 0;
	int* dev_arr_2 = 0;

	hipSetDevice(0);
	hipMalloc((void**)&dev_ans, size * sizeof(int));
	hipMalloc((void**)&dev_arr_1, size * sizeof(int));
	hipMalloc((void**)&dev_arr_2, size * sizeof(int));

	hipMemcpy(dev_arr_1, arr_1, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr_2, arr_2, size * sizeof(int), hipMemcpyHostToDevice);

	if (mood == 's') {
		k_vector_sum_vector << <ceil(size / 500), 500 >> > (dev_ans, dev_arr_1, dev_arr_2, size);
	}
	else {
		k_vector_minus_vector << <ceil(size / 500), 500 >> > (dev_ans, dev_arr_1, dev_arr_2, size);
	}
	hipDeviceSynchronize();

	hipMemcpy(ans, dev_ans, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_ans);
	hipFree(dev_arr_1);
	hipFree(dev_arr_2);
	return;
}

// Programando CPU <-- Separando ambos ambientes //
int main() {
	int size = 1000;
	int* arr = new int[size];
	int* arr_2 = new int[size];
	int* answer = new int[size];
	int constant = 2; // Cantidad a sumar o restar

	for (int i = 0; i < size; i++) {//Inicializando
		arr[i] = i;
		arr_2[i] = i + i;
		answer[i] = 1;
	}

	//Probando suma resta de vector con constante
	//cout << "Primer vector" << endl;
	//print_function(arr, size);
	//facade_k_vector_sum_minus_constante(arr, answer, size, constant, 'm'); // s -> suma, m -> resta
	//cout << "Vector resultante" << endl;
	//print_function(answer, size);

	//Probando suma resta de vector con vector 
	cout << "Primer vector" << endl;
	print_function(arr, size);
	cout << "Segundo vector" << endl;
	print_function(arr_2, size);
	facade_k_vector_sum_minus_vector(answer, arr, arr_2, size, 's'); // s -> suma, m -> resta
	cout << "Vector resultante" << endl;
	print_function(answer, size);

	hipDeviceReset();
	return 0;
}

void print_function(int* arr, int size) {
	for (int i = 0; i < size; i++) {
		cout << arr[i] << ", ";
	}
	cout << endl << endl;
}



/*
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/