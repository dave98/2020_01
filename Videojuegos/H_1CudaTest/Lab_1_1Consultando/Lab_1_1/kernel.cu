#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

void main() {
	hipDeviceProp_t prop;
	int count = 0;
	hipGetDeviceCount(&count);

	for (int index = 0; index < count; index++) {
		hipGetDeviceProperties(&prop, index);

		cout << prop.name << endl;
		cout << prop.minor << " - " << prop.major << endl;
		cout << prop.clockRate << endl;
	}
}

/* Resultados */
/*
GeForce 930MX
0 - 5
1019500
*/