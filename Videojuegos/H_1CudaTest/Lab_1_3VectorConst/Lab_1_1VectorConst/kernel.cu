#include "hip/hip_runtime.h"



#include <stdio.h>
#include <iostream>	
using namespace std;

void print_function(int*, int);

// Programando Tarjeta// 
__global__ void k_vector_x_constante(int* arr, int* answer, int n, int k) { // arr -> Vector, n -> tama�o de array, k flotante
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		answer[idx] = arr[idx] * k;
	}
}


void facade_k_vector_x_constante(int *arr, int* answer, unsigned int size, int k) {
	int *dev_arr = 0;
	int *dev_answer = 0;
	int dev_k = k;

	//cudaSetDevice(0); // Seleccionando primera tarjeta de video en caso de sistemas con multiples tarjetas.
	
	hipMalloc((void**)&dev_arr, size * sizeof(int)); // Reservando espacio de memoria en dispositivo
	hipMalloc((void**)&dev_answer, size * sizeof(int));

	hipMemcpy(dev_arr, arr, size * sizeof(int), hipMemcpyHostToDevice); // Copiando contenido de vector a dispositivo
	//k_vector_x_constante<<<ceil(size / 512), 512>>>(dev_arr, dev_answer, size, dev_k); // <<<n_blocks, n_threads >>>
	k_vector_x_constante <<< ceil(size / 500), 500 >>> (dev_arr, dev_answer, size, dev_k); // Can run many blocks  at the same time, newer gps 1024 threads, older gpu 512
	hipDeviceSynchronize();
	

	hipMemcpy(answer, dev_answer, size * sizeof(int), hipMemcpyDeviceToHost); // Devolviendo datos del dispositivo
	hipFree(dev_arr);
	hipFree(dev_answer);
	return;
}
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Programando CPU <-- Separando ambos ambientes //
int main() {
	int size = 10000;
	int* arr = new int[size];
	int* answer = new int[size];
	int constant = 2;

	for (int i = 0; i < size; i++) {//Inicializando
		arr[i] = i;
		answer[i] = 1;
	}

	print_function(arr, size);
	facade_k_vector_x_constante(arr, answer, size, constant);
	print_function(answer, size);
	
	hipDeviceReset();
	return 0;
}

void print_function(int* arr, int size) {
	for (int i = 0; i < size; i++) {
		cout << arr[i] << ", ";
	}
	cout << endl << endl;
}
