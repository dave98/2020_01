#include "hip/hip_runtime.h"



#include <stdio.h>
#include <iostream>
using namespace std;

// Can�t use iostream enviroments inside cuda kernels
__global__ void myFirstKernel() {
	printf("First kernel");
}

int main() {
	myFirstKernel <<< 1, 1 >>> ();
}